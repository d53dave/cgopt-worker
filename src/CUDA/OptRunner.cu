#include "hip/hip_runtime.h"
//
// Created by dotdi on 20.11.16.
//

#include "OptRunner.h"


void CSAOpt::OptRunner::simulated_annealing(size_t rounds, size_t nThreads, Target *results, hiprandState *states,
                                            Optimization::RandomDistr distribution) {
    int id = threadIdx.x;

    OptimizationImpl opt;

    printf("Thread %d started.\n", id);

    double (*rand_func)(hiprandStateXORWOW_t *state);

    switch(distribution){
        case Optimization::normal:
            rand_func = &hiprand_normal_double;
            break;
        case Optimization::uniform:
        default:
            rand_func = &hiprand_uniform_double;
            break;
    }

    printf("Rand_func initialized\n");

    double rands[RAND_SIZE];
    for (int i = 0; i < RAND_SIZE; ++i) {
        rands[i] = rand_func(&states[id]);
    }

    printf("%d rands generated\n", RAND_SIZE);

    //Target &state, double *rands
    auto target = results[id];
    auto cur_state = opt.initialize(target, rands);

    printf("Cur state calculated\n");
    auto state_best = cur_state;
    auto energy_old = opt.evaluate(cur_state);
    auto energy_best = energy_old;
    double temp = opt.cool(rounds); // TODO: return type should be configurable here

    printf("Initial state initialized e=%f\n", energy_best);

    for(size_t round = rounds; temp >= 0.0 && --round >= 0;){

        for (int k = 0; k < RAND_SIZE; ++k) {
            rands[k] = rand_func(&states[id]);
        }

        auto state_new = opt.generateNext(cur_state, rands);
        auto energy_new = opt.evaluate(state_new);

        if(energy_new < energy_best){
            state_best =   state_new ;
            energy_best =  energy_new ;
            state_best.energy = energy_new;
        }

        temp = opt.temp_func(round);

        if( energy_new < energy_old || exp( (energy_old - energy_new) / temp ) > hiprand_uniform(&states[id]) ){
            printf("New step to state with e=%f at temp %f\n", energy_new, temp);
            cur_state = state_new;
            energy_old = energy_new;
        }
    }
    results[id] = state_best;
}

__global__
void CSAOpt::OptRunner::setup_kernel(hiprandState *state, unsigned long seed) {
    int id = threadIdx.x;
    hiprand_init( seed, id, 0, &state[id]);
}

void CSAOpt::OptRunner::run_simulated_annealing() {
    size_t nThreads = 10;

    Target* targets;
    checkCudaErrors(hipMallocManaged(&targets, sizeof(Target)*nThreads));

    for(size_t i=0; i<nThreads; ++i){
        Target t;
        for(size_t k=0; k<RAND_SIZE; ++k){
            t.coords[k] = 0.1*k;
        }
        targets[i] = t;
    }

    hiprandState* devStates;
    checkCudaErrors(hipMallocManaged(&devStates, sizeof(hiprandState)*nThreads));

    printf("Running cuRand setup_kernel... ");

    setup_kernel<<<1, nThreads>>>(devStates, time(NULL));

    printf("\t\tdone.\n");
    printf("Running simulated annealing... \n");
    //Optimization *opt, size_t rounds, size_t nThreads, Target *result, Optimization::RandomDistr distr

    simulated_annealing<<<1, nThreads>>>(10000, nThreads, targets, devStates, Optimization::uniform);


    hipDeviceSynchronize();
    print_arr(targets, nThreads, "results");

    checkCudaErrors(hipFree(devStates));
    return (0);
}